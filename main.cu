
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cmath>
#include <ctime>

using namespace std;

static void HandleError( hipError_t err,
                         const char *file,
                         int line );
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

void cudaCheckAndPrintProperties();
hipError_t addWithCuda(double* c, const double* a, const double* b, unsigned int size, const int BLOCKS, const int THREADS_PER_BLOCK);
void printTest();

__global__ void add(int a, int b, int* c) {
    *c = a + b;
}
double fRand(double fMin, double fMax);

const extern int ARRAY_SIZE = 6000000;
const extern int NORMAL_SPREAD = 12;

int main() {
    cudaCheckAndPrintProperties();

    printf("Array size: %d", ARRAY_SIZE);
    const int BLOCKS = (!ARRAY_SIZE % 1024) ? ARRAY_SIZE / 1024 : ARRAY_SIZE / 1024 + 1;


    const int THREADS_PER_BLOCK = (!ARRAY_SIZE % BLOCKS) ? 1024 : ARRAY_SIZE / BLOCKS +
                                                                  ceil(double(ARRAY_SIZE % BLOCKS) / BLOCKS);

    printf("\nAmount of BLOCKS: %d",  BLOCKS);
    printf("\nTHREADS_PER_BLOCK: %d",  THREADS_PER_BLOCK);

    auto * a = (double *)calloc(ARRAY_SIZE, sizeof(double ));
    auto * b = (double *)calloc(ARRAY_SIZE, sizeof(double ));
    auto * c = (double *)calloc(ARRAY_SIZE, sizeof(double ));


    srand(time(nullptr));
    // инициализация

    for (int i = 0; i < ARRAY_SIZE; ++i)
    {
        a[i] = fRand(0, 10);
        b[i] = fRand(10, 20);
    }



    // Пареллельное сложения на GPU
    if (addWithCuda(c, a, b, ARRAY_SIZE, BLOCKS, THREADS_PER_BLOCK) != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        exit( EXIT_FAILURE );
    }


    for (int i = 0; i < 3; ++i){
        printf("\n%d: %f + %f = %f", i, a[i], b[i], c[i]);
    }
    for (int i = ARRAY_SIZE - 3; i < ARRAY_SIZE; ++i){
        printf("\n%d: %f + %f = %f", i, a[i], b[i], c[i]);
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    if (hipDeviceReset() != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        exit( EXIT_FAILURE );
    }

    free(a);
    free(b);
    free(c);

    return EXIT_SUCCESS;

}


void printTest(){
    cout <<"\n\n\n";
    cout<< "TTTTTTTTTTTTTTTTTTTTTTTEEEEEEEEEEEEEEEEEEEEEE   SSSSSSSSSSSSSSS TTTTTTTTTTTTTTTTTTTTTTT\n"
           "T:::::::::::::::::::::TE::::::::::::::::::::E SS:::::::::::::::ST:::::::::::::::::::::T\n"
           "T:::::::::::::::::::::TE::::::::::::::::::::ES:::::SSSSSS::::::ST:::::::::::::::::::::T\n"
           "T:::::TT:::::::TT:::::TEE::::::EEEEEEEEE::::ES:::::S     SSSSSSST:::::TT:::::::TT:::::T\n"
           "TTTTTT  T:::::T  TTTTTT  E:::::E       EEEEEES:::::S            TTTTTT  T:::::T  TTTTTT\n"
           "        T:::::T          E:::::E             S:::::S                    T:::::T        \n"
           "        T:::::T          E::::::EEEEEEEEEE    S::::SSSS                 T:::::T        \n"
           "        T:::::T          E:::::::::::::::E     SS::::::SSSSS            T:::::T        \n"
           "        T:::::T          E:::::::::::::::E       SSS::::::::SS          T:::::T        \n"
           "        T:::::T          E::::::EEEEEEEEEE          SSSSSS::::S         T:::::T        \n"
           "        T:::::T          E:::::E                         S:::::S        T:::::T        \n"
           "        T:::::T          E:::::E       EEEEEE            S:::::S        T:::::T        \n"
           "      TT:::::::TT      EE::::::EEEEEEEE:::::ESSSSSSS     S:::::S      TT:::::::TT      \n"
           "      T:::::::::T      E::::::::::::::::::::ES::::::SSSSSS:::::S      T:::::::::T      \n"
           "      T:::::::::T      E::::::::::::::::::::ES:::::::::::::::SS       T:::::::::T      \n"
           "      TTTTTTTTTTT      EEEEEEEEEEEEEEEEEEEEEE SSSSSSSSSSSSSSS         TTTTTTTTTTT      \n" << endl;
    cout <<
            "     CCC::::::::::::CU::::::U     U::::::UD::::::::::::DDD                   A:::A                   \n"
            "   CC:::::::::::::::CU::::::U     U::::::UD:::::::::::::::DD                A:::::A                  \n"
            "  C:::::CCCCCCCC::::CUU:::::U     U:::::UUDDD:::::DDDDD:::::D              A:::::::A                 \n"
            " C:::::C       CCCCCC U:::::U     U:::::U   D:::::D    D:::::D            A:::::::::A                \n"
            "C:::::C               U:::::D     D:::::U   D:::::D     D:::::D          A:::::A:::::A               \n"
            "C:::::C               U:::::D     D:::::U   D:::::D     D:::::D         A:::::A A:::::A              \n"
            "C:::::C               U:::::D     D:::::U   D:::::D     D:::::D        A:::::A   A:::::A             \n"
            "C:::::C               U:::::D     D:::::U   D:::::D     D:::::D       A:::::A     A:::::A            \n"
            "C:::::C               U:::::D     D:::::U   D:::::D     D:::::D      A:::::AAAAAAAAA:::::A           \n"
            "C:::::C               U:::::D     D:::::U   D:::::D     D:::::D     A:::::::::::::::::::::A          \n"
            " C:::::C       CCCCCC U::::::U   U::::::U   D:::::D    D:::::D     A:::::AAAAAAAAAAAAA:::::A         \n"
            "  C:::::CCCCCCCC::::C U:::::::UUU:::::::U DDD:::::DDDDD:::::D     A:::::A             A:::::A        \n"
            "   CC:::::::::::::::C  UU:::::::::::::UU  D:::::::::::::::DD     A:::::A               A:::::A       \n"
            "     CCC::::::::::::C    UU:::::::::UU    D::::::::::::DDD      A:::::A                 A:::::A      \n"
            "        CCCCCCCCCCCCC      UUUUUUUUU      DDDDDDDDDDDDD        AAAAAAA                   AAAAAAA\n";
}

void cudaCheckAndPrintProperties(){
    printTest();
    cout << "------------------------------------------------------------------------------------------------\n";
    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));

    cout << "Found " << count << " device(s)" << endl;

    for (int i = 0; i < count; ++i) {
        hipDeviceProp_t prop{};

        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
        cout << "Device name: " << prop.name << endl;
        cout << "Warp size in threads: " << prop.warpSize << endl;
        cout << "Shared memory available per block in bytes: " << prop.sharedMemPerBlock  / 1024.0 / 1024.0 << " MB" << endl;
        cout << "Total Memory: " << prop.totalGlobalMem / 1024.0 / 1024.0 << " MB" << endl;
        cout << "Max Threads per Block: " << prop.maxThreadsPerBlock << endl;
        cout << "Maximum size of each dimension of a grid: " <<  prop.maxGridSize[0] << " | "
                                                             <<  prop.maxGridSize[1] << " | "
                                                             <<  prop.maxGridSize[2] << endl;

        cout << "Maximum size of each dimension of a block: " <<  prop.maxThreadsDim[0] << " | "
                                                              <<  prop.maxThreadsDim[1] << " | "
                                                              <<  prop.maxThreadsDim[2] << endl;
        cout << "Compute capability: " << prop.major << "." << prop.minor << endl;
    }

    int result;
    int* devResult;

    HANDLE_ERROR(hipMalloc((void**)&devResult, sizeof(int)));

    add<<<1, 1>>>(7, 8, devResult);

    HANDLE_ERROR(hipMemcpy(&result, devResult, sizeof(int), hipMemcpyDeviceToHost));

    cout << "7 + 8 = " << result << endl;
    hipFree(devResult);
    cout << "------------------------------------------------------------------------------------------------\n";
}


//__global__ — выполняется на GPU, вызывается с CPU.
__global__ void addKernel(double *c, const double *a, const double *b, const int size)
{
    // Индекс обсчитываемых компонент вектора с учетом смещения от количества блоков
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size)
        c[i] = a[i] + b[i];
}

hipError_t addWithCuda(double* c, const double* a, const double* b, unsigned int size, const int BLOCKS, const int THREADS_PER_BLOCK)
{
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    double allTime = 0;
    hipError_t cudaStatus;

    // Создание обработчиков событий
    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Cannot create CUDA start event: %s\n",
                hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Cannot create CUDA end event: %s\n",
                hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Инициализация девайса
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    // Выделения памяти на GPU
    cudaStatus = hipMalloc(&dev_c, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc(&dev_a, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc(&dev_b, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Копирования входных векторов с хоста на девайс
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    for (int i = 0; i < 12; i++)
    {
        // Установка точки старта
        cudaStatus = hipEventRecord(start, 0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Cannot record CUDA start event: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // Запуск функции ядра на GPU
        addKernel <<< BLOCKS, THREADS_PER_BLOCK >>> (dev_c, dev_a, dev_b, size);

        // Отлов ошибок запуска ядра
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // Установка точки окончания
        cudaStatus = hipEventRecord(stop, 0);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "Cannot record CUDA end event: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // Ожидание завершения обсчета функции ядра
        // Отлов ошибок работы и завершения ядра
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto Error;
        }

        // Расчет времени
        cudaStatus = hipEventElapsedTime(&gpuTime, start, stop);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "Cannot record CUDA time event: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }
        printf("\nTime: %.20f", gpuTime / 1000);
        allTime += gpuTime / 1000;
    }
    printf("\nAverage time: %.20f", allTime / 12);

    // Копирования выходного вектора с девайса на хост
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Возникла ошибка/конец программы
    Error:
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}



void HandleError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

double fRand(double fMin, double fMax) {
    double f = (double)rand() / RAND_MAX;
    return fMin + f * (fMax - fMin);
}
